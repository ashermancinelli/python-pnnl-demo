
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "hipblas.h"

void add_vec_kernel(double* a, double* b, double* c, int size)
{
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  double *da, *db;
  
  cudaStat = hipMalloc((void**)&da, sizeof(double) * size); assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&db, sizeof(double) * size); assert(cudaStat == hipSuccess);
  cudaStat = hipMemcpy(da, a, sizeof(double) * size, hipMemcpyHostToDevice); assert(cudaStat == hipSuccess);
  cudaStat = hipMemcpy(db, b, sizeof(double) * size, hipMemcpyHostToDevice); assert(cudaStat == hipSuccess);
  stat = hipblasCreate(&handle); assert(stat == HIPBLAS_STATUS_SUCCESS);
  double one=1.;
  stat = hipblasDaxpy(handle, size, &one, a, 1, b, 1); assert(stat == HIPBLAS_STATUS_SUCCESS);
  cudaStat = hipMemcpy(c, db, sizeof(double) * size, hipMemcpyDeviceToHost); assert(cudaStat == hipSuccess);
  stat = hipblasDestroy(handle);
  hipFree(da);
  hipFree(db);
}
